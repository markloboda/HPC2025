#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

// STB image library
#define STB_IMAGE_IMPLEMENTATION
#include "lib/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image_write.h"

// Constants
#define HISTOGRAM_LEVELS 256
#define COLOR_CHANNELS 3

// Settings
#define SAVE_TIMING_STATS
#define WRITE_OUTPUT_IMAGE

// Macros
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define CLAMP(a, min, max) ((a) < (min) ? (min) : ((a) > (max) ? (max) : (a)))
#define CLAMP255(a) CLAMP(a, 0, 255)

struct execution_result
{
    int width;
    int height;
    float hist;
    float cdf;
    float equalize;
    float sum;
    float total;
};

unsigned int findMin(unsigned int *cdf)
{
    unsigned int min = 0;
    for (int i = 0; min == 0 && i < HISTOGRAM_LEVELS; i++)
    {
        min = cdf[i];
    }
    return min;
}

unsigned char scale(unsigned int cdf, unsigned int cdfmin, unsigned int imageSize)
{
    int scale = CLAMP255(floor(((float)(cdf - cdfmin) / (float)(imageSize - cdfmin)) * (HISTOGRAM_LEVELS - 1.0)));
    return (unsigned char) scale;
}

void RGBtoYUV(unsigned char *image, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            unsigned int pixelIdx = (y * width + x) * COLOR_CHANNELS;

            float r = (float)image[pixelIdx + 0];
            float g = (float)image[pixelIdx + 1];
            float b = (float)image[pixelIdx + 2];

            // YUV conversion formula
            unsigned char y = (unsigned char) CLAMP255((    0.299f * r +    0.587f * g +    0.114f * b) +   0.0f);
            unsigned char u = (unsigned char) CLAMP255((-0.168736f * r - 0.331264f * g +      0.5f * b) + 128.0f);
            unsigned char v = (unsigned char) CLAMP255((      0.5f * r - 0.418688f * g - 0.081312f * b) + 128.0f);

            // assign YUV values back to the image
            image[pixelIdx + 0] = y;
            image[pixelIdx + 1] = u;
            image[pixelIdx + 2] = v;
        }
    }
}

void CalculateHistogram(unsigned char *image, int width, int height, unsigned int *histogram)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            unsigned int pixelIdx = (y * width + x) * COLOR_CHANNELS;
            histogram[image[pixelIdx]]++;
        }
    }
}

void CalculateCDF(unsigned int *histogram, unsigned int *cdf)
{
    cdf[0] = histogram[0];
    for (int i = 1; i < HISTOGRAM_LEVELS; i++)
    {
        cdf[i] = cdf[i - 1] + histogram[i];
    }
}

void Equalize(unsigned char *image, int width, int height, unsigned int *cdf)
{
    unsigned int imageSize = width * height;
    unsigned int cdfmin = findMin(cdf);

    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            unsigned int pixelIdx = (y * width + x) * COLOR_CHANNELS;
            image[pixelIdx] = scale(cdf[image[pixelIdx]], cdfmin, imageSize);
        }
    }
}

void YUVtoRGB(unsigned char *image, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            unsigned int pixelIdx = (y * width + x) * COLOR_CHANNELS;

            float y = (float)image[pixelIdx + 0];
            float u = (float)image[pixelIdx + 1];
            float v = (float)image[pixelIdx + 2];

            // RBG conversion formula
            u -= 128.0f;
            v -= 128.0f;

            unsigned char r = (unsigned char) CLAMP255((1.0f * y +      0.0f * u +    1.402f * v));
            unsigned char g = (unsigned char) CLAMP255((1.0f * y - 0.344136f * u - 0.714136f * v));
            unsigned char b = (unsigned char) CLAMP255((1.0f * y +    1.772f * u +      0.0f * v));

            // assign YUV values back to the image
            image[pixelIdx + 0] = r;
            image[pixelIdx + 1] = g;
            image[pixelIdx + 2] = b;
        }
    }
}

int main(int argc, char *args[])
{
    if (argc != 3)
    {
        printf("Error: Invalid amount of arguments. [%d]\n", argc);
        exit(1);
    }

    char *imageInPath = args[1];
    char *imageOutPath = args[2];

    // Read image from file
    int imageWidthPixel, imageHeightPixel, cpp;
    unsigned char *image = stbi_load(imageInPath, &imageWidthPixel, &imageHeightPixel, &cpp, COLOR_CHANNELS);
    if (image == NULL)
    {
        printf("Error in loading the image\n");
        return EXIT_FAILURE;
    }
    if (cpp != COLOR_CHANNELS)
    {
        printf("Error: Image is not RGB\n");
        return EXIT_FAILURE;
    }

    // Allocate memory for raw output image data, histogram, and CDF
    unsigned int *histogram = (unsigned int *) calloc(HISTOGRAM_LEVELS, sizeof(unsigned int));
    unsigned int *CDF = (unsigned int *) calloc(HISTOGRAM_LEVELS, sizeof(unsigned int));

    // Create time events
    hipEvent_t startMain, stopMain,
                startTimeRGBtoYUV, stopTimeRGBtoYUV, 
                startTimeHistogramMS, stopTimeHistogramMS, 
                startTimeCumulativeMS, stopTimeCumulativeMS, 
                startTimeEqualizeMS, stopTimeEqualizeMS, 
                startTimeYUVtoRGB, stopTimeYUVtoRGB;

    hipEventCreate(&startMain);
    hipEventCreate(&stopMain);
    hipEventCreate(&startTimeRGBtoYUV);
    hipEventCreate(&stopTimeRGBtoYUV);
    hipEventCreate(&startTimeHistogramMS);
    hipEventCreate(&stopTimeHistogramMS);
    hipEventCreate(&startTimeCumulativeMS);
    hipEventCreate(&stopTimeCumulativeMS);
    hipEventCreate(&startTimeEqualizeMS);
    hipEventCreate(&stopTimeEqualizeMS);
    hipEventCreate(&startTimeYUVtoRGB);
    hipEventCreate(&stopTimeYUVtoRGB);

    float elapsedTimeRGBtoYUV = 0,
          elapsedTimeHistogramMS= 0,
          elapsedTimeCumulativeMS= 0, 
          elapsedTimeEqualizeMS= 0,
          elapsedTimeYUVtoRGB= 0,
          elapsedMain= 0;

    hipEventRecord(startMain);

    // 1. Transform the image from RGB to YUV
    hipEventRecord(startTimeRGBtoYUV);
    RGBtoYUV(image, imageWidthPixel, imageHeightPixel);
    hipEventRecord(stopTimeRGBtoYUV);

    // 2. Compute the luminance histogram
    hipEventRecord(startTimeHistogramMS);
    CalculateHistogram(image, imageWidthPixel, imageHeightPixel, histogram);
    hipEventRecord(stopTimeHistogramMS);

    // 3. Calculate the cumulative histogram
    hipEventRecord(startTimeCumulativeMS);
    CalculateCDF(histogram, CDF);
    hipEventRecord(stopTimeCumulativeMS);

    // 4. Calculate new pixel luminances from original luminance based on the histogram equalization formula
    // 5. Assign new luminance to each pixel
    hipEventRecord(startTimeEqualizeMS);
    Equalize(image, imageWidthPixel, imageHeightPixel, CDF);
    hipEventRecord(stopTimeEqualizeMS);

    // 6. Convert the image back to RGB colour space
    hipEventRecord(startTimeYUVtoRGB);
    YUVtoRGB(image, imageWidthPixel, imageHeightPixel);
    hipEventRecord(stopTimeYUVtoRGB);

    // End the time recording and calculate elapsed times
    hipEventRecord(stopMain);
    hipEventSynchronize(stopMain);

    hipEventElapsedTime(&elapsedMain, startMain, stopMain);
    hipEventElapsedTime(&elapsedTimeRGBtoYUV, startTimeRGBtoYUV, stopTimeRGBtoYUV);
    hipEventElapsedTime(&elapsedTimeHistogramMS, startTimeHistogramMS, stopTimeHistogramMS);
    hipEventElapsedTime(&elapsedTimeCumulativeMS, startTimeCumulativeMS, stopTimeCumulativeMS);
    hipEventElapsedTime(&elapsedTimeEqualizeMS, startTimeEqualizeMS, stopTimeEqualizeMS);
    hipEventElapsedTime(&elapsedTimeYUVtoRGB, startTimeYUVtoRGB, stopTimeYUVtoRGB);
    
    elapsedTimeHistogramMS += elapsedTimeRGBtoYUV; // add RGB to YUV time to compare with CUDA implementation
    elapsedTimeEqualizeMS += elapsedTimeYUVtoRGB; // add YUV to RGB time to compare with CUDA implementation


// Output timing stats to file //////////////////////////////////////////////////////////////////////////
#ifdef SAVE_TIMING_STATS
    struct execution_result result;
    result.width = imageWidthPixel;
    result.height = imageHeightPixel;
    result.hist = elapsedTimeHistogramMS;
    result.cdf = elapsedTimeCumulativeMS;
    result.equalize = elapsedTimeEqualizeMS;
    result.sum = elapsedTimeHistogramMS + elapsedTimeCumulativeMS + elapsedTimeEqualizeMS;
    result.total = elapsedMain;

    FILE *timingFile = fopen("./timing_stats/timing_stats_serial.txt", "a");
    fprintf(timingFile, "--------------- HISTOGRAM EQUALIZATION - Serial ---------------\n");
    fprintf(timingFile, "--------------- %s ---------------\n", imageInPath);
    fprintf(timingFile, "Image width: %d\n", imageWidthPixel);
    fprintf(timingFile, "Image height: %d\n", imageHeightPixel);
    fprintf(timingFile, "Histogram: %f ms\n", result.hist);
    fprintf(timingFile, "CDF: %f ms\n", result.cdf);
    fprintf(timingFile, "Equalize: %f ms\n", result.equalize);
    fprintf(timingFile, "Total time: %f ms\n", result.total);
    fprintf(timingFile, "Sum of all times: %f ms\n", result.sum);
    fprintf(timingFile, "-----------------------------------------------------\n");
    fprintf(timingFile, "\n");
    fclose(timingFile);
#endif

#ifdef WRITE_OUTPUT_IMAGE
    // Write output image:
    stbi_write_png(imageOutPath, imageWidthPixel, imageHeightPixel, COLOR_CHANNELS, image, imageWidthPixel * COLOR_CHANNELS);
#endif

    // Clean-up events
    hipEventDestroy(startMain);
    hipEventDestroy(stopMain);
    hipEventDestroy(startTimeRGBtoYUV);
    hipEventDestroy(stopTimeRGBtoYUV);
    hipEventDestroy(startTimeHistogramMS);
    hipEventDestroy(stopTimeHistogramMS);
    hipEventDestroy(startTimeCumulativeMS);
    hipEventDestroy(stopTimeCumulativeMS);
    hipEventDestroy(startTimeEqualizeMS);
    hipEventDestroy(stopTimeEqualizeMS);
    hipEventDestroy(startTimeYUVtoRGB);
    hipEventDestroy(stopTimeYUVtoRGB);

    // Free memory
    stbi_image_free(image);
    free(histogram);
    free(CDF);

    return EXIT_SUCCESS;
}