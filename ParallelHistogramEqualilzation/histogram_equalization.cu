#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// STB image library
#define STB_IMAGE_IMPLEMENTATION
#include "lib/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image_write.h"

// Constants
#define HISTOGRAM_LEVELS 256
#define COLOR_CHANNELS 3

// Settings
#define SAVE_TIMING_STATS
#define WRITE_OUTPUT_IMAGE

// Macros
#define ELAPSED_TIME_MS(start, stop) (stop - start) / (double)CLOCKS_PER_SEC * 1000
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define CLAMP(a, min, max) ((a) < (min) ? (min) : ((a) > (max) ? (max) : (a)))
#define CLAMP255(a) CLAMP(a, 0, 255)

struct execution_result
{
    int width;
    int height;
    float hist;
    float cdf;
    float equalize;
    float sum;
    float total;
};

unsigned int findMin(unsigned int *cdf)
{
    unsigned int min = 0;
    for (int i = 0; min == 0 && i < HISTOGRAM_LEVELS; i++)
    {
        min = cdf[i];
    }
    return min;
}

unsigned char scale(unsigned int cdf, unsigned int cdfmin, unsigned int imageSize)
{
    int scale = CLAMP255(floor(((float)(cdf - cdfmin) / (float)(imageSize - cdfmin)) * (HISTOGRAM_LEVELS - 1.0)));
    return (unsigned char) scale;
}

void RGBtoYUV(unsigned char *image, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            unsigned int pixelIdx = (y * width + x) * COLOR_CHANNELS;

            float r = (float)image[pixelIdx + 0];
            float g = (float)image[pixelIdx + 1];
            float b = (float)image[pixelIdx + 2];

            // YUV conversion formula
            unsigned char y = (unsigned char) CLAMP255((    0.299f * r +    0.587f * g +    0.114f * b) +   0.0f);
            unsigned char u = (unsigned char) CLAMP255((-0.168736f * r - 0.331264f * g +      0.5f * b) + 128.0f);
            unsigned char v = (unsigned char) CLAMP255((      0.5f * r - 0.418688f * g - 0.081312f * b) + 128.0f);

            // assign YUV values back to the image
            image[pixelIdx + 0] = y;
            image[pixelIdx + 1] = u;
            image[pixelIdx + 2] = v;
        }
    }
}

void CalculateHistogram(unsigned char *image, int width, int height, unsigned int *histogram)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            unsigned int pixelIdx = (y * width + x) * COLOR_CHANNELS;
            histogram[image[pixelIdx]]++;
        }
    }
}

void CalculateCDF(unsigned int *histogram, unsigned int *cdf)
{
    cdf[0] = histogram[0];
    for (int i = 1; i < HISTOGRAM_LEVELS; i++)
    {
        cdf[i] = cdf[i - 1] + histogram[i];
    }
}

void Equalize(unsigned char *image, int width, int height, unsigned int *cdf)
{
    unsigned int imageSize = width * height;
    unsigned int cdfmin = findMin(cdf);

    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            unsigned int pixelIdx = (y * width + x) * COLOR_CHANNELS;
            image[pixelIdx] = scale(cdf[image[pixelIdx]], cdfmin, imageSize);
        }
    }
}

void YUVtoRGB(unsigned char *image, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            unsigned int pixelIdx = (y * width + x) * COLOR_CHANNELS;

            float y = (float)image[pixelIdx + 0];
            float u = (float)image[pixelIdx + 1];
            float v = (float)image[pixelIdx + 2];

            // RBG conversion formula
            u -= 128.0f;
            v -= 128.0f;

            unsigned char r = (unsigned char) CLAMP255((1.0f * y +      0.0f * u +    1.402f * v));
            unsigned char g = (unsigned char) CLAMP255((1.0f * y - 0.344136f * u - 0.714136f * v));
            unsigned char b = (unsigned char) CLAMP255((1.0f * y +    1.772f * u +      0.0f * v));

            // assign YUV values back to the image
            image[pixelIdx + 0] = r;
            image[pixelIdx + 1] = g;
            image[pixelIdx + 2] = b;
        }
    }
}

int main(int argc, char *args[])
{
    if (argc != 3)
    {
        printf("Error: Invalid amount of arguments. [%d]\n", argc);
        exit(1);
    }

    char *imageInPath = args[1];
    char *imageOutPath = args[2];

    // Read image from file
    int imageWidthPixel, imageHeightPixel, cpp;
    unsigned char *image = stbi_load(imageInPath, &imageWidthPixel, &imageHeightPixel, &cpp, COLOR_CHANNELS);
    if (image == NULL)
    {
        printf("Error in loading the image\n");
        return EXIT_FAILURE;
    }
    if (cpp != COLOR_CHANNELS)
    {
        printf("Error: Image is not RGB\n");
        return EXIT_FAILURE;
    }

    // Allocate memory for raw output image data, histogram, and CDF
    unsigned int *histogram = (unsigned int *) calloc(HISTOGRAM_LEVELS, sizeof(unsigned int));
    unsigned int *CDF = (unsigned int *) calloc(HISTOGRAM_LEVELS, sizeof(unsigned int));

    clock_t startMain, stopMain;
    startMain = clock();

    float elapsedTimeRGBtoYUV = 0,
          elapsedTimeHistogramMS= 0,
          elapsedTimeCumulativeMS= 0, 
          elapsedTimeEqualizeMS= 0,
          elapsedTimeYUVtoRGB= 0,
          elapsedMain= 0;

    clock_t start, stop;
    // 1. Transform the image from RGB to YUV
    start = clock();
    RGBtoYUV(image, imageWidthPixel, imageHeightPixel);
    stop = clock();
    elapsedTimeRGBtoYUV = ELAPSED_TIME_MS(start, stop);

    // 2. Compute the luminance histogram
    start = clock();
    CalculateHistogram(image, imageWidthPixel, imageHeightPixel, histogram);
    stop = clock();
    elapsedTimeHistogramMS = ELAPSED_TIME_MS(start, stop) + elapsedTimeRGBtoYUV; // add RGB to YUV time to compare with CUDA implementation

    // 3. Calculate the cumulative histogram
    start = clock();
    CalculateCDF(histogram, CDF);
    stop = clock();
    elapsedTimeCumulativeMS = ELAPSED_TIME_MS(start, stop);

    // 4. Calculate new pixel luminances from original luminance based on the histogram equalization formula
    // 5. Assign new luminance to each pixel
    start = clock();
    Equalize(image, imageWidthPixel, imageHeightPixel, CDF);
    stop = clock();
    elapsedTimeEqualizeMS = ELAPSED_TIME_MS(start, stop);

    // 6. Convert the image back to RGB colour space
    start = clock();
    YUVtoRGB(image, imageWidthPixel, imageHeightPixel);
    stop = clock();
    elapsedTimeYUVtoRGB = ELAPSED_TIME_MS(start, stop);

    elapsedTimeEqualizeMS += elapsedTimeYUVtoRGB; // add YUV to RGB time to compare with CUDA implementation

    stopMain = clock();
    elapsedMain = ELAPSED_TIME_MS(startMain, stopMain);

// Output timing stats to file //////////////////////////////////////////////////////////////////////////
#ifdef SAVE_TIMING_STATS
    struct execution_result result;
    result.width = imageWidthPixel;
    result.height = imageHeightPixel;
    result.hist = elapsedTimeHistogramMS;
    result.cdf = elapsedTimeCumulativeMS;
    result.equalize = elapsedTimeEqualizeMS;
    result.sum = elapsedTimeHistogramMS + elapsedTimeCumulativeMS + elapsedTimeEqualizeMS;
    result.total = elapsedMain;

    FILE *timingFile = fopen("./timing_stats/timing_stats_serial.txt", "a");
    fprintf(timingFile, "--------------- HISTOGRAM EQUALIZATION - Serial ---------------\n");
    fprintf(timingFile, "--------------- %s ---------------\n", imageInPath);
    fprintf(timingFile, "Image width: %d\n", imageWidthPixel);
    fprintf(timingFile, "Image height: %d\n", imageHeightPixel);
    fprintf(timingFile, "Histogram: %f ms\n", result.hist);
    fprintf(timingFile, "CDF: %f ms\n", result.cdf);
    fprintf(timingFile, "Equalize: %f ms\n", result.equalize);
    fprintf(timingFile, "Total time: %f ms\n", result.total);
    fprintf(timingFile, "Sum of all times: %f ms\n", result.sum);
    fprintf(timingFile, "-----------------------------------------------------\n");
    fprintf(timingFile, "\n");
    fclose(timingFile);
#endif

#ifdef WRITE_OUTPUT_IMAGE
    // Write output image:
    stbi_write_png(imageOutPath, imageWidthPixel, imageHeightPixel, COLOR_CHANNELS, image, imageWidthPixel * COLOR_CHANNELS);
#endif

    // Free memory
    stbi_image_free(image);
    free(histogram);
    free(CDF);

    return EXIT_SUCCESS;
}