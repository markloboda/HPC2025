#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// STB image library
#define STB_IMAGE_IMPLEMENTATION
#include "lib/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image_write.h"

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "lib/hip/hip_runtime_api.h"

// Settings
#define HISTOGRAM_LEVELS 256
#define COLOR_CHANNELS 3
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define SAVE_TIMING_STATS

// Macros
#define ELAPSED_TIME_MS(start, stop) (stop - start) / (double)CLOCKS_PER_SEC * 1000
#define CLAMP(a, min, max) ((a) < (min) ? (min) : ((a) > (max) ? (max) : (a)))
#define CLAMP255(a) CLAMP(a, 0, 255)
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

void calculateHistogram(unsigned char *image, int imageWidthPixel, int imageHeightPixel, int imageSizeBytes, unsigned int *histogram);
__global__ void calculateHistogram_kernel(unsigned char *imageData, const int imageWidth, const int imageHeight, unsigned int *sharedHistogram);

void calculateCumulativeDistibution(unsigned int *histogram, unsigned int *cumulativeDistributionHistogram);
__global__ void calculateCumulativeDistribution_kernel(unsigned int *deviceInHistogram, unsigned int *deviceOutHistogram, int histogramSize);

void equalize(unsigned char *imageIn, unsigned char *imageOut, int imageWidthPixel, int imageHeightPixel, int imageSizeBytes, unsigned int *cumulativeDistributionHistogram);
__global__ void equalize_kernel(unsigned char *deviceImageIn, unsigned char *deviceImageOut, int imageWidthPixel, int imageHeightPixel, int threadIdOffset, unsigned int *cdfmin, unsigned int *deviceCumulativeDistributionHistogram);
__global__ void findMin_kernel(unsigned int *deviceCumulativeDistributionHistogram, unsigned int *minimum);
__device__ inline unsigned char scale_device(unsigned int cdf, unsigned int cdfmin, unsigned int imageSize);

void printHistogram(unsigned int *histogram);
void printKernelRuntime(float elapsedTimeMS);

float elapsedTimeHistogramMS, elapsedTimeCumulativeMS, elapsedTimeEqualizeMS;
struct hipDeviceProp_t props;

struct execution_result
{
    int width;
    int height;
    float hist;
    float cdf;
    float equalize;
    float sum;
    float total;
};

int main(int argc, char *args[])
{
    if (argc != 3)
    {
        printf("Error: Invalid amount of arguments. [%d]\n", argc);
        exit(1);
    }

    char *imageInPath = args[1];
    char *imageOutPath = args[2];

    ///// load image
    int imageWidthPixel, imageHeightPixel, cpp, imageSizeBytes;
    unsigned char *image = stbi_load(imageInPath, &imageWidthPixel, &imageHeightPixel, &cpp, COLOR_CHANNELS);
    if (image == NULL)
    {
        printf("Error: Couldn't load image\n");
        exit(1);
    }
    if (cpp != COLOR_CHANNELS)
    {
        printf("Error: Image is not RGB\n");
        return 1;
    }

    imageSizeBytes = imageWidthPixel * imageHeightPixel * COLOR_CHANNELS * sizeof(unsigned char);

    int device;
    hipGetDeviceProperties(&props, hipGetDevice(&device));

    clock_t startMain, stopMain;
    startMain = clock();

    ////// STEP 1: Image to YUV and compute the histogram
    unsigned int *histogram = (unsigned int *)malloc(HISTOGRAM_LEVELS * sizeof(unsigned int));
    calculateHistogram(image, imageWidthPixel, imageHeightPixel, imageSizeBytes, histogram);

    ////// STEP 2: Compute the cumulative distribution of the histogram
    unsigned int *cumulativeDistributionHistogram = (unsigned int *)malloc(HISTOGRAM_LEVELS * sizeof(unsigned int));
    calculateCumulativeDistibution(histogram, cumulativeDistributionHistogram);

    ////// STEP 3: Transform the original image using the scaled cumulative distribution as the transformation function
    equalize(image, image, imageWidthPixel, imageHeightPixel, imageSizeBytes, cumulativeDistributionHistogram);

    stopMain = clock();
    float elapsedTimeMain = ELAPSED_TIME_MS(startMain, stopMain);

// Output timing stats to file //////////////////////////////////////////////////////////////////////////
#ifdef SAVE_TIMING_STATS
    // execution stats
    struct execution_result result;
    result.width = imageWidthPixel;
    result.height = imageHeightPixel;
    result.hist = elapsedTimeHistogramMS;
    result.cdf = elapsedTimeCumulativeMS;
    result.equalize = elapsedTimeEqualizeMS;
    result.sum = elapsedTimeHistogramMS + elapsedTimeCumulativeMS + elapsedTimeEqualizeMS;
    result.total = elapsedTimeMain;
    write(STDOUT_FILENO, &result, sizeof(struct execution_result));

    FILE *timingFile = fopen("./timing_stats/timing_stats_parallel.txt", "a");
    fprintf(timingFile, "--------------- HISTOGRAM EQUALIZATION - Parallel ---------------\n", imageInPath);
    fprintf(timingFile, "--------------- %s ---------------\n", imageInPath);
    fprintf(timingFile, "Image width: %d\n", imageWidthPixel);
    fprintf(timingFile, "Image height: %d\n", imageHeightPixel);
    fprintf(timingFile, "Histogram: %f ms\n", result.hist);
    fprintf(timingFile, "CDF: %f ms\n", result.cdf);
    fprintf(timingFile, "Equalize: %f ms\n", result.equalize);
    fprintf(timingFile, "Total time: %f ms\n", result.total);
    fprintf(timingFile, "Sum of all times: %f ms\n", result.sum);
    fprintf(timingFile, "-----------------------------------------------------\n");
    fprintf(timingFile, "\n");
    fclose(timingFile);
#endif

    // write output image:
    stbi_write_png(imageOutPath, imageWidthPixel, imageHeightPixel, COLOR_CHANNELS, image, imageWidthPixel * COLOR_CHANNELS);

    stbi_image_free(image);
    free(image);
    free(histogram);
    free(cumulativeDistributionHistogram);

    return 0;
}

void calculateHistogram(unsigned char *image, int imageWidthPixel, int imageHeightPixel, int imageSizeBytes, unsigned int *histogram)
{
    // pointer to the data of the image on the GPU
    unsigned char *deviceImage;
    hipMalloc((void **)&deviceImage, imageSizeBytes);
    hipMemcpy(deviceImage, image, imageSizeBytes, hipMemcpyHostToDevice);
    // pointer to the histogram on the GPU
    unsigned int *deviceHistogram;
    hipMalloc((void **)&deviceHistogram, HISTOGRAM_LEVELS * sizeof(unsigned int));
    hipMemset(deviceHistogram, 0, HISTOGRAM_LEVELS * sizeof(unsigned int));
    getLastCudaError("setting up GPU data faled in: calculateHistogram()");

    // set up the grid and block size
    dim3 gridSize(ceil(imageWidthPixel * imageHeightPixel / (float)HISTOGRAM_LEVELS));
    dim3 blockSize(HISTOGRAM_LEVELS);

    // create timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // runs KERNEL
    calculateHistogram_kernel<<<gridSize, blockSize>>>(deviceImage, imageWidthPixel, imageHeightPixel, deviceHistogram);
    getLastCudaError("calculateHistogram_kernel() execution failed");

    // get elaspedTime
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeMS;
    hipEventElapsedTime(&elapsedTimeMS, start, stop);
    getLastCudaError("calculating elapsed time failed in calculateHistogram() failed");

    // recover data from the GPU to the CPU allocated memory
    hipMemcpy(image, deviceImage, imageSizeBytes, hipMemcpyDeviceToHost);
    hipMemcpy(histogram, deviceHistogram, HISTOGRAM_LEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    getLastCudaError("retrieving data from GPU failed in: calculateHistogram()");

    // /////// output:
    // printf("---------HISTOGRAM--------\n");
    // printKernelRuntime(elapsedTimeMS);
    // printf("--------------------------\n");
    // printHistogram(histogram);
    // printf("--------------------------\n");

    hipFree(deviceImage);
    hipFree(deviceHistogram);
    getLastCudaError("freeing memory in calculateHistogram() failed");

    elapsedTimeHistogramMS = elapsedTimeMS;
}

__global__ void calculateHistogram_kernel(unsigned char *imageData, const int imageWidth, const int imageHeight, unsigned int *sharedHistogram)
{
    __shared__ unsigned int blockHistogram[HISTOGRAM_LEVELS];

    // reset the value of the gray value
    // TODO: test without as cudamemset is called.
    blockHistogram[threadIdx.x] = 0;

    __syncthreads();

    // find index of the pixel of the thread
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int indexOffset = blockDim.x * gridDim.x;

    // check current y levels and increment corresponding values
    int imagePixelSize = imageWidth * imageHeight;
    while (index < imagePixelSize)
    {
        unsigned int pixelIdx = index * COLOR_CHANNELS;

        // RBG to YUV conversion
        float r = (float)imageData[pixelIdx + 0];
        float g = (float)imageData[pixelIdx + 1];
        float b = (float)imageData[pixelIdx + 2];
        imageData[pixelIdx + 0] = (unsigned char) CLAMP255((    0.299f * r +    0.587f * g +    0.114f * b));
        imageData[pixelIdx + 1] = (unsigned char) CLAMP255((-0.168736f * r - 0.331264f * g +      0.5f * b) + 128.0f);
        imageData[pixelIdx + 2] = (unsigned char) CLAMP255((      0.5f * r - 0.418688f * g - 0.081312f * b) + 128.0f);

        atomicAdd(&blockHistogram[imageData[pixelIdx]], 1);
        index += indexOffset;
    }

    __syncthreads();

    // add the calculated value of the thread to the main shared histogram
    atomicAdd(&sharedHistogram[threadIdx.x], blockHistogram[threadIdx.x]);
}

void calculateCumulativeDistibution(unsigned int *histogram, unsigned int *cumulativeDistributionHistogram)
{
    // pointer to the input histogram on the GPU
    unsigned int *deviceInHistogram;
    hipMalloc((void **)&deviceInHistogram, HISTOGRAM_LEVELS * sizeof(unsigned int));
    hipMemcpy(deviceInHistogram, histogram, HISTOGRAM_LEVELS * sizeof(unsigned int), hipMemcpyHostToDevice);
    // pointer to the output histogram on the GPU
    unsigned int *deviceOutHistogram;
    hipMalloc((void **)&deviceOutHistogram, HISTOGRAM_LEVELS * sizeof(unsigned int));
    getLastCudaError("setting up GPU data faled in: calculateCumulativeDistibution()");

    // set up the grid and block size
    dim3 gridSize(1);
    dim3 blockSize(HISTOGRAM_LEVELS);

    // create timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // runs KERNEL
    calculateCumulativeDistribution_kernel<<<gridSize, blockSize>>>(deviceInHistogram, deviceOutHistogram, HISTOGRAM_LEVELS);
    getLastCudaError("calculateCumulativeDistribution_kernel() execution failed");

    // get elaspedTime
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeMS;
    hipEventElapsedTime(&elapsedTimeMS, start, stop);
    getLastCudaError("calculating elapsed time in calculateCumulativeDistribution() failed");

    // recover data from the GPU to the CPU allocated memory
    hipMemcpy(cumulativeDistributionHistogram, deviceOutHistogram, HISTOGRAM_LEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    getLastCudaError("retrieving data from GPU failed in: calculateCumulativeDistribution()");

    // /////// output:
    // printf("------------CDF-----------\n");
    // printKernelRuntime(elapsedTimeMS);
    // printf("--------------------------\n");
    // printHistogram(cumulativeDistributionHistogram);
    // printf("--------------------------\n");

    hipFree(deviceInHistogram);
    hipFree(deviceOutHistogram);
    getLastCudaError("freeing memory in calculateCumulativeDistribution() failed");

    elapsedTimeCumulativeMS = elapsedTimeMS;
}

// algorithm explained: [https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda]
__global__ void calculateCumulativeDistribution_kernel(unsigned int *deviceInHistogram, unsigned int *deviceOutHistogram, int histogramSize)
{
    __shared__ unsigned int temp[HISTOGRAM_LEVELS * sizeof(unsigned int)];
    int tid = threadIdx.x;
    int offset = 1;

    // a
    int ai = tid;
    int bi = tid + (histogramSize / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    temp[ai + bankOffsetA] = deviceInHistogram[ai];
    temp[bi + bankOffsetB] = deviceInHistogram[bi];

    for (int d = histogramSize >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (tid < d)
        {
            // b
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    // c
    int lastElement;
    if (tid == 0)
    {
        lastElement = temp[histogramSize - 1 + CONFLICT_FREE_OFFSET(histogramSize - 1)];
        temp[histogramSize - 1 + CONFLICT_FREE_OFFSET(histogramSize - 1)] = 0;
    }

    for (int d = 1; d < histogramSize; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (tid < d)
        {
            // d
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    // e
    deviceOutHistogram[ai - 1] = temp[ai + bankOffsetA];
    deviceOutHistogram[bi - 1] = temp[bi + bankOffsetB];

    if (tid == 0)
        deviceOutHistogram[histogramSize - 1] = lastElement;
}

void equalize(unsigned char *imageIn, unsigned char *imageOut, int imageWidthPixel, int imageHeightPixel, int imageSizeBytes, unsigned int *cumulativeDistributionHistogram)
{
    // pointer to the image input on the GPU
    unsigned char *deviceImageIn;
    hipMalloc((void **)&deviceImageIn, imageSizeBytes);
    hipMemcpy(deviceImageIn, imageIn, imageSizeBytes, hipMemcpyHostToDevice);

    // pointer to the image output on the GPU
    unsigned char *deviceImageOut;
    hipMalloc((void **)&deviceImageOut, imageSizeBytes);

    // pointer to the cumulative distribution histogram on the GPU
    unsigned int *deviceCumulativeDistributionHistogram;
    hipMalloc((void **)&deviceCumulativeDistributionHistogram, HISTOGRAM_LEVELS * sizeof(unsigned int));
    hipMemcpy(deviceCumulativeDistributionHistogram, cumulativeDistributionHistogram, HISTOGRAM_LEVELS * sizeof(unsigned int), hipMemcpyHostToDevice);

    // pointer to the non zero minimum in the cumulative distribution on the GPU
    unsigned int *cdfmin;
    hipMalloc((void **)&cdfmin, sizeof(unsigned int));
    getLastCudaError("setting up GPU data faled in: equalize()");

    dim3 gridSizeMin(1);
    dim3 blockSizeMin(HISTOGRAM_LEVELS);

    findMin_kernel<<<gridSizeMin, blockSizeMin>>>(deviceCumulativeDistributionHistogram, cdfmin);
    getLastCudaError("findMin_kernel() execution failed");

    dim3 gridSizeEqualize(ceil(imageWidthPixel * imageHeightPixel) / 256.0);
    dim3 blockSizeEqualize(256);

    // pointer to the thread id offset on new iteration
    int threadIdOffset = blockSizeEqualize.x * gridSizeEqualize.x;

    // create events meant for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    equalize_kernel<<<gridSizeEqualize, blockSizeEqualize>>>(deviceImageIn, deviceImageOut, imageWidthPixel, imageHeightPixel, threadIdOffset, cdfmin, deviceCumulativeDistributionHistogram);
    getLastCudaError("equalize_kernel() execution failed");

    // get elaspedTime
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeMS;
    hipEventElapsedTime(&elapsedTimeMS, start, stop);
    getLastCudaError("calculating elapsed time in equalize() failed");

    // recover data from the GPU to the CPU allocated memory
    hipMemcpy(imageOut, deviceImageOut, imageSizeBytes, hipMemcpyDeviceToHost);
    getLastCudaError("retrieving data from GPU failed in: equalize()");

    hipFree(deviceImageIn);
    hipFree(deviceImageOut);
    hipFree(deviceCumulativeDistributionHistogram);
    getLastCudaError("freeing memory in equalize() failed");

    elapsedTimeEqualizeMS = elapsedTimeMS;
}

__global__ void equalize_kernel(unsigned char *deviceImageIn, unsigned char *deviceImageOut, int imageWidthPixel, int imageHeightPixel, int threadIdOffset, unsigned int *cdfmin, unsigned int *deviceCumulativeDistributionHistogram)
{
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;

    while (threadId < imageWidthPixel * imageHeightPixel)
    {
        unsigned int pixelIdx = threadId * COLOR_CHANNELS;

        // YUV to RGB conversion
        float y = scale_device(deviceCumulativeDistributionHistogram[deviceImageIn[pixelIdx]], *cdfmin, imageWidthPixel * imageHeightPixel);
        float u = (float)deviceImageIn[pixelIdx + 1] - 128.0f;
        float v = (float)deviceImageIn[pixelIdx + 2] - 128.0f;

        deviceImageOut[pixelIdx + 0] = (unsigned char)(CLAMP255((float)(y + 1.402f * v)));
        deviceImageOut[pixelIdx + 1] = (unsigned char)(CLAMP255((float)(y - 0.344136f * u - 0.714136f * v)));
        deviceImageOut[pixelIdx + 2] = (unsigned char)(CLAMP255((float)(y + 1.772f * u)));

        threadId += threadIdOffset;
    }
}

__global__ void findMin_kernel(unsigned int *deviceCumulativeDistributionHistogram, unsigned int *minimum)
{
    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (threadIdx.x < i)
        {
            if (deviceCumulativeDistributionHistogram[threadIdx.x + 1] == 0 && deviceCumulativeDistributionHistogram[threadIdx.x] == 0)
            {
                deviceCumulativeDistributionHistogram[threadIdx.x] = UINT32_MAX;
            }
            else
            {

                deviceCumulativeDistributionHistogram[threadIdx.x] =
                    deviceCumulativeDistributionHistogram[threadIdx.x + 1] < deviceCumulativeDistributionHistogram[threadIdx.x] && deviceCumulativeDistributionHistogram[threadIdx.x + 1] != 0
                        ? deviceCumulativeDistributionHistogram[threadIdx.x + 1]
                        : deviceCumulativeDistributionHistogram[threadIdx.x];
            }
        }
        i /= 2;
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        *minimum = deviceCumulativeDistributionHistogram[0];
    }
}

__device__ inline unsigned char scale_device(unsigned int cdf, unsigned int cdfmin, unsigned int imageSize)
{
    int scale = CLAMP255(floor(((float)(cdf - cdfmin) / (float)(imageSize - cdfmin)) * (HISTOGRAM_LEVELS - 1.0)));
    return (unsigned char)scale;
}

void printHistogram(unsigned int *histogram)
{
    for (int i = 0; i < HISTOGRAM_LEVELS; i++)
    {
        printf("%i = %llu\n", i, histogram[i]);
    }
}

void printKernelRuntime(float elapsedTimeMS)
{
    printf("Kerner run time: %3.3f ms\n", elapsedTimeMS);
}